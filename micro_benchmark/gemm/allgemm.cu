
#include <hip/hip_runtime.h>
#include <iostream>
#include <hiprand/hiprand.h>
#include <hipblas.h>
#include <iomanip>
#include <vector>
#include <cstdlib>

#define MAX(x, y) ((x>y) ? x : y)
// Define some error checking macros.
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
	if (stat != hipSuccess) {
		fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
	}
}

#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
	}
}

#define curandErrCheck(stat) { curandErrCheck_((stat), __FILE__, __LINE__); }
void curandErrCheck_(hiprandStatus_t stat, const char *file, int line) {
	if (stat != HIPRAND_STATUS_SUCCESS) {
		fprintf(stderr, "cuRand Error: %d %s %d\n", stat, file, line);
	}
}


double cal_tflops(int m, int n, int k, double msec)
{
    double flops = 2. * m * n * k;
    double tflops = (1E-12*flops) / (1E-3*msec);
    return tflops;
}

 

__global__ void assignFloatValue (float *out, int n, float value) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < n) {
		out[idx] = value;
	}
}

__global__ void assignHalfValue (half *out, int n, float value) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < n) {
		out[idx] = value;
	}
}
void correctnessCheck(int m, int n, int k, float *host, float value){
        for (int i = 0; i < m * n; i++) {      
            float val = host[i];
            if ( val != k * value * value) {
                std::cout << "ERROR value = " << val<< std::endl;
            }
        }
}

void printTime(float cublasTime, int m, int n, int k, float &s_max_tflops, int &s_max_m_n, int &s_max_k ){
        float tflops = cal_tflops(m, n, k, cublasTime);
        if (tflops > s_max_tflops){
            s_max_tflops = tflops;
	    s_max_m_n = m;
            s_max_k = k;
        }
        std::cout << std::setw(7) << m << ",";
        std::cout << std::setw(7) << n << ",";
        std::cout << std::setw(7) << k << ",";
        std::cout << std::setw(15) << std::setprecision(4) << cublasTime << ",";
        std::cout << std::setw(15) << std::setprecision(4) << tflops << "," << std::endl;
}

void calFP16Tensor(int m, int n, int k, float &s_max_tflops, int &s_max_m_n, int &s_max_k, int numRepeats){
        half *a_fp16;
        half *b_fp16;
        half *c_cublas;
        float *c_host_cublas;
        const float  value = 1.0f;
   
        hipblasHandle_t cublasHandle;

        hipEvent_t startcublas;
        hipEvent_t stopcublas;

        cudaErrCheck(hipEventCreate(&startcublas));
        cudaErrCheck(hipEventCreate(&stopcublas));
        cublasErrCheck(hipblasCreate(&cublasHandle));
        // Use tensor cores
        cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));

        cudaErrCheck(hipMalloc((void**)&a_fp16, m * k * sizeof(half)));
        cudaErrCheck(hipMalloc((void**)&b_fp16, k * n * sizeof(half)));
        cudaErrCheck(hipMalloc((void**)&c_cublas, m * n * sizeof(half)));
        c_host_cublas = (float*)malloc(m * n * sizeof(float));

        // curand doesn't currently support fp16 so we generate in fp32 and convert to fp16.
        assignHalfValue <<< (m * k + 255) / 256, 256 >>> (a_fp16, m*k, value);
        assignHalfValue <<< (k * n + 255) / 256, 256 >>> (b_fp16, k*n, value);
        assignHalfValue <<< (k * n + 255) / 256, 256 >>> (c_cublas, m*n, 0.0f);

        float alpha = 1.0f;
        float beta = 0.0f;

        // Now using cuBLAS
        cudaErrCheck(hipEventRecord(startcublas));
        for (int iteration = 0; iteration < numRepeats; ++iteration) {
        hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, 
                    m, n, k, 
                    &alpha,
                    a_fp16, HIP_R_16F, m,
                    b_fp16, HIP_R_16F, n,
                    &beta, 
                    c_cublas, HIP_R_16F, m,
                    HIP_R_16F, CUBLAS_GEMM_DFALT_TENSOR_OP);
        }
        cudaErrCheck(hipEventRecord(stopcublas));
        cudaErrCheck(hipEventSynchronize(stopcublas));
        // TODO: Correctness check
        //cudaErrCheck(cudaMemcpy(c_host_cublas, c_cublas, m * n * sizeof(float), cudaMemcpyDeviceToHost));
        //correctnessCheck(m, n, k, c_host_cublas, value);
        // Check time
        float cublasTime;	
        cudaErrCheck(hipEventElapsedTime(&cublasTime, startcublas, stopcublas)); 
        cublasTime /= numRepeats;
        printTime(cublasTime, m, n, k, s_max_tflops, s_max_m_n, s_max_k);
        
        cudaErrCheck(hipEventDestroy(startcublas));             
        cudaErrCheck(hipEventDestroy(stopcublas));
        cudaErrCheck(hipFree(a_fp16));
        cudaErrCheck(hipFree(b_fp16));
        cudaErrCheck(hipFree(c_cublas));
        free(c_host_cublas);
}

void calFP16Accu32Tensor(int m, int n, int k, float &s_max_tflops, int &s_max_m_n, int &s_max_k, int numRepeats){
        half *a_fp16;
        half *b_fp16;
        float *c_cublas;
        float *c_host_cublas;
        const float  value = 1.0f;
        hipblasHandle_t cublasHandle;
        hipEvent_t startcublas;
        hipEvent_t stopcublas;

        cudaErrCheck(hipEventCreate(&startcublas));
        cudaErrCheck(hipEventCreate(&stopcublas));
        cublasErrCheck(hipblasCreate(&cublasHandle));
        // Use tensor cores
        cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));

        cudaErrCheck(hipMalloc((void**)&a_fp16, m * k * sizeof(half)));
        cudaErrCheck(hipMalloc((void**)&b_fp16, k * n * sizeof(half)));
        cudaErrCheck(hipMalloc((void**)&c_cublas, m * n * sizeof(float)));
        c_host_cublas = (float*)malloc(m * n * sizeof(float));

        // curand doesn't currently support fp16 so we generate in fp32 and convert to fp16.
        assignHalfValue <<< (m * k + 255) / 256, 256 >>> (a_fp16, m*k, value);
        assignHalfValue <<< (k * n + 255) / 256, 256 >>> (b_fp16, k*n, value);
        assignFloatValue <<< (k * n + 255) / 256, 256 >>> (c_cublas, m*n, 0.0f);

        float alpha = 1.0f;
        float beta = 0.0f;
        // Warp up not really needed
        // Now using cuBLAS
        cudaErrCheck(hipEventRecord(startcublas));
        for (int iteration = 0; iteration < numRepeats; ++iteration) {
        cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, 
                    m, n, k, 
                    &alpha,
                    a_fp16, HIP_R_16F, m,
                    b_fp16, HIP_R_16F, n,
                    &beta, 
                    c_cublas, HIP_R_32F, m,
                    HIP_R_32F, CUBLAS_GEMM_DFALT_TENSOR_OP));
        }
        cudaErrCheck(hipEventRecord(stopcublas));
        cudaErrCheck(hipEventSynchronize(stopcublas));
        // Correctness check
        cudaErrCheck(hipMemcpy(c_host_cublas, c_cublas, m * n * sizeof(float), hipMemcpyDeviceToHost));
        correctnessCheck(m, n, k, c_host_cublas, value);
        // Check time
        float cublasTime;	
        cudaErrCheck(hipEventElapsedTime(&cublasTime, startcublas, stopcublas)); 
        cublasTime /= numRepeats;
        printTime(cublasTime, m, n, k, s_max_tflops, s_max_m_n, s_max_k);
        
        cudaErrCheck(hipEventDestroy(startcublas));             
        cudaErrCheck(hipEventDestroy(stopcublas));
        cudaErrCheck(hipFree(a_fp16));
        cudaErrCheck(hipFree(b_fp16));
        cudaErrCheck(hipFree(c_cublas));
        free(c_host_cublas);
}

void calFP32CUDA(int m, int n, int k, float &s_max_tflops, int &s_max_m_n, int &s_max_k, int numRepeats){
        float *a_fp32;
        float *b_fp32;
        float *c_cublas;
        float *c_host_cublas;
        const float  value = 1.0f;
        hipblasHandle_t cublasHandle;

        hipEvent_t startcublas;
        hipEvent_t stopcublas;

        cudaErrCheck(hipEventCreate(&startcublas));
        cudaErrCheck(hipEventCreate(&stopcublas));
        cublasErrCheck(hipblasCreate(&cublasHandle));
        // No tensor cores
        cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_DEFAULT_MATH));

        cudaErrCheck(hipMalloc((void**)&a_fp32, m * k * sizeof(float)));
        cudaErrCheck(hipMalloc((void**)&b_fp32, k * n * sizeof(float)));
        cudaErrCheck(hipMalloc((void**)&c_cublas, m * n * sizeof(float)));
        c_host_cublas = (float*)malloc(m * n * sizeof(float));

        // curand doesn't currently support fp16 so we generate in fp32 and convert to fp16.
        assignFloatValue <<< (m * k + 255) / 256, 256 >>> (a_fp32, m*k, value);
        assignFloatValue <<< (k * n + 255) / 256, 256 >>> (b_fp32, k*n, value);
        assignFloatValue <<< (k * n + 255) / 256, 256 >>> (c_cublas, m*n, 0.0f);

        float alpha = 1.0f;
        float beta = 0.0f;
        
        cudaErrCheck(hipEventRecord(startcublas));
        for (int iteration = 0; iteration < numRepeats; ++iteration) {
        hipblasSgemm(cublasHandle,
                HIPBLAS_OP_N,
                HIPBLAS_OP_T,
                m,
                n,
                k,
                &alpha,
                a_fp32, m,
                b_fp32, n,
                &beta,
                c_cublas, m);
        }
        cudaErrCheck(hipEventRecord(stopcublas));
        cudaErrCheck(hipEventSynchronize(stopcublas));
        // Correctness check
        cudaErrCheck(hipMemcpy(c_host_cublas, c_cublas, m * n * sizeof(float), hipMemcpyDeviceToHost));
        correctnessCheck(m, n, k, c_host_cublas, value);
        // Check time
        float cublasTime = 0.0f;	
        cudaErrCheck(hipEventElapsedTime(&cublasTime, startcublas, stopcublas)); 
        cublasTime /= numRepeats;
        printTime(cublasTime, m, n, k, s_max_tflops, s_max_m_n, s_max_k);
        
        cudaErrCheck(hipEventDestroy(startcublas));             
        cudaErrCheck(hipEventDestroy(stopcublas));
        cudaErrCheck(hipFree(a_fp32));
        cudaErrCheck(hipFree(b_fp32));
        cudaErrCheck(hipFree(c_cublas));
        free(c_host_cublas);
}


void calFP16CUDA(int m, int n, int k, float &s_max_tflops, int &s_max_m_n, int &s_max_k, int numRepeats){
        half *a_fp16;
        half *b_fp16;
        half *c_cublas;
        float *c_host_cublas;
        const float  value = 1.0f;
   
        hipblasHandle_t cublasHandle;

        hipEvent_t startcublas;
        hipEvent_t stopcublas;

        cudaErrCheck(hipEventCreate(&startcublas));
        cudaErrCheck(hipEventCreate(&stopcublas));
        cublasErrCheck(hipblasCreate(&cublasHandle));
        // No tensor cores
        cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_DEFAULT_MATH));

        cudaErrCheck(hipMalloc((void**)&a_fp16, m * k * sizeof(half)));
        cudaErrCheck(hipMalloc((void**)&b_fp16, k * n * sizeof(half)));
        cudaErrCheck(hipMalloc((void**)&c_cublas, m * n * sizeof(half)));
        c_host_cublas = (float*)malloc(m * n * sizeof(float));

        // curand doesn't currently support fp16 so we generate in fp32 and convert to fp16.
        assignHalfValue <<< (m * k + 255) / 256, 256 >>> (a_fp16, m*k, value);
        assignHalfValue <<< (k * n + 255) / 256, 256 >>> (b_fp16, k*n, value);
        assignHalfValue <<< (k * n + 255) / 256, 256 >>> (c_cublas, m*n, 0.0f);

        half alpha = 1.0f;
        half beta = 0.0f;

        // Now using cuBLAS
        cudaErrCheck(hipEventRecord(startcublas));
        for (int iteration = 0; iteration < numRepeats; ++iteration) {
        hipblasHgemm(cublasHandle,
                HIPBLAS_OP_N,
                HIPBLAS_OP_T,
                m,
                n,
                k,
                &alpha,
                a_fp16, m,
                b_fp16, n,
                &beta,
                c_cublas, m);
        }
        cudaErrCheck(hipEventRecord(stopcublas));
        cudaErrCheck(hipEventSynchronize(stopcublas));
        // TODO: Correctness check
        //cudaErrCheck(cudaMemcpy(c_host_cublas, c_cublas, m * n * sizeof(float), cudaMemcpyDeviceToHost));
       //correctnessCheck(m, n, k, c_host_cublas, value);
        // Check time
        float cublasTime;	
        cudaErrCheck(hipEventElapsedTime(&cublasTime, startcublas, stopcublas)); 
        cublasTime /= numRepeats;
        printTime(cublasTime, m, n, k, s_max_tflops, s_max_m_n, s_max_k);
        
        cudaErrCheck(hipEventDestroy(startcublas));             
        cudaErrCheck(hipEventDestroy(stopcublas));
        cudaErrCheck(hipFree(a_fp16));
        cudaErrCheck(hipFree(b_fp16));
        cudaErrCheck(hipFree(c_cublas));
        free(c_host_cublas);
}
int main(int argc, char* argv[]) {
    int m,n,k;
    std::string precision="NULL";
    bool perf = true;
    if (argc < 3) {
        return EXIT_FAILURE;
    }
    
    // precision = INT8_TENSOR
    // precision = FP16_TENSOR
    // precision = FP16_32_TENSOR
    // precision = FP32_CUDA
    // precision = FP16_CUDA
    if (argc == 3) {
        precision = argv[1];
        std::string tmp = argv[2];
        if (tmp == "performance") perf= true;
	else if (tmp == "pressure") perf = false;
	else {
	  std::cout << "Invalid parameters!"<<std::endl;
	  return EXIT_FAILURE;
	}
    }
    
    float s_max_tflops = 0;
    int s_max_m_n = 0;
    int s_max_k = 0;
    int numRepeats;
/* // deprecated this INT8 test as it will achieve the best perf. Please refer to cublasLt    
    if (precision == "INT8_TENSOR" || precision == "NULL") {
    std::cout << "[TensorCore INT8(INT32 accumulation) Time and TOPS Result]" << std::endl;
    std::cout << std::setw(7) << "m" << std::setw(7) << "n" << std::setw(7) << "k";
    std::cout << std::setw(15) << "Time (msec)" << std::setw(15) << "TOPS";
    std::cout << std::endl;
    
    // for tensorcore test TODO: to verify the int8 with int8 accumulation
    for(m=1024, n = 1024; m <= 25600; m+=1024, n+=1024) {
    for(k=1024; k <= 20480; k+=1024) {
  
        int8_t *a_;
        int8_t *b_;
        int *c_cublas;
        int *c_host_cublas;
        //const int  value = 1;

   
        cublasHandle_t cublasHandle;

        cudaEvent_t startcublas;
        cudaEvent_t stopcublas;

        cudaErrCheck(cudaEventCreate(&startcublas));
        cudaErrCheck(cudaEventCreate(&stopcublas));
        cublasErrCheck(cublasCreate(&cublasHandle));
        // Use tensor cores
        cublasErrCheck(cublasSetMathMode(cublasHandle, CUBLAS_TENSOR_OP_MATH));

        cudaErrCheck(cudaMalloc((void**)&a_, m * k * sizeof(int8_t)));
        cudaErrCheck(cudaMalloc((void**)&b_, k * m * sizeof(int8_t)));
        cudaErrCheck(cudaMalloc((void**)&c_cublas, m * n * sizeof(int)));
        c_host_cublas = (int*)malloc(m * n * sizeof(int));

        //TODO curand doesn't currently support fp16 so we generate in fp32 and convert to fp16.
        //assignHalfValue <<< (m * k + 255) / 256, 256 >>> (a_fp16, m*k, value);
        //assignHalfValue <<< (k * n + 255) / 256, 256 >>> (b_fp16, k*n, value);
        //assignHalfValue <<< (k * n + 255) / 256, 256 >>> (c_cublas, m*n, 0.0f);

        int alpha = 1;
        int beta = 0;
        int numRepeats = 1;
        // Warp up not really needed here as many params will be tested
        // Now using cuBLAS
        cudaErrCheck(cudaEventRecord(startcublas));
        for (int iteration = 0; iteration < numRepeats; ++iteration) {
        cublasErrCheck(cublasGemmEx(cublasHandle, CUBLAS_OP_N, CUBLAS_OP_T, 
                    m, n, k, 
                    &alpha,
                    a_, CUDA_R_8I, m,
                    b_, CUDA_R_8I, n,
                    &beta, 
                    c_cublas, CUDA_R_32I, m,
                    CUDA_R_32I, CUBLAS_GEMM_DFALT_TENSOR_OP));
        }
        cudaErrCheck(cudaEventRecord(stopcublas));
        cudaErrCheck(cudaEventSynchronize(stopcublas));
        // TODO: Correctness check
        //cudaErrCheck(cudaMemcpy(c_host_cublas, c_cublas, m * n * sizeof(float), cudaMemcpyDeviceToHost));
       //correctnessCheck(m, n, k, c_host_cublas, value);
        // Check time
        float cublasTime;	
        cudaErrCheck(cudaEventElapsedTime(&cublasTime, startcublas, stopcublas)); 
        cublasTime /= numRepeats;
        printTime(cublasTime, m, n, k, s_max_tflops, s_max_m_n, s_max_k);
        
        cudaErrCheck(cudaEventDestroy(startcublas));             
        cudaErrCheck(cudaEventDestroy(stopcublas));
        cudaErrCheck(cudaFree(a_));
        cudaErrCheck(cudaFree(b_));
        cudaErrCheck(cudaFree(c_cublas));
        free(c_host_cublas);
    }}
    std::cout << "[Peak TFLOPS]=" << s_max_tflops << ", m=n="<< s_max_m_n << ", k="<<s_max_k<< std::endl;
    cudaErrCheck(cudaDeviceReset());
    }
*/  

    //======= for tensorcore test
    // for perf test
    if (precision == "FP16_TENSOR" && perf == true) {
    std::cout << "[TensorCore FP16(FP16 accumulation) Time and TFLOPS Result]" << std::endl;
    std::cout << std::setw(7) << "m" << std::setw(7) << "n" << std::setw(7) << "k";
    std::cout << std::setw(15) << "Time (msec)" << std::setw(15) << "TFLOPS";
    std::cout << std::endl;
    s_max_tflops = 0;
    s_max_m_n = 0;
    s_max_k = 0;
    numRepeats = 10;
    for(m=1024, n = 1024; m <= 25600; m+=4096, n+=4096) {
    for(k=1024; k <= 20480; k+=4096) {
	calFP16Tensor( m, n, k,s_max_tflops, s_max_m_n, s_max_k, numRepeats);
    }}
    std::cout << "[Peak TFLOPS]=" << s_max_tflops << ", m=n="<< s_max_m_n << ", k="<<s_max_k<< std::endl;
    cudaErrCheck(hipDeviceReset());
    }
    
    // for pressure test
    if (precision == "FP16_TENSOR" && perf == false) {
    std::cout << "[TensorCore FP16(FP16 accumulation) Time and TFLOPS Result]" << std::endl;
    std::cout << std::setw(7) << "m" << std::setw(7) << "n" << std::setw(7) << "k";
    std::cout << std::setw(15) << "Time (msec)" << std::setw(15) << "TFLOPS";
    std::cout << std::endl;
    s_max_tflops = 0;
    s_max_m_n = 0;
    s_max_k = 0;
    numRepeats = 2000;
    std::vector<int> mnk={512, 1024, 5120, 10240};
    for(int i=0; i<mnk.size(); i++) calFP16Tensor( mnk[i], mnk[i], mnk[i], s_max_tflops, s_max_m_n, s_max_k, numRepeats);
    
    cudaErrCheck(hipDeviceReset());
    }
 
    // for perf test
    if (precision == "FP16_32_TENSOR" && perf == true) {
    std::cout << "[TensorCore FP16(FP32 accumulation) Time and TFLOPS Result]" << std::endl;
    std::cout << std::setw(7) << "m" << std::setw(7) << "n" << std::setw(7) << "k";
    std::cout << std::setw(15) << "Time (msec)" << std::setw(15) << "TFLOPS";
    std::cout << std::endl;
    s_max_tflops = 0;
    numRepeats = 10;
    for(m=1024, n = 1024; m <= 25600; m+=4096, n+=4096) {
    for(k=1024; k <= 20480; k+=4096) {
  	calFP16Accu32Tensor( m, n, k, s_max_tflops, s_max_m_n, s_max_k, numRepeats);
    }}
    std::cout << "[Peak TFLOPS]=" << s_max_tflops << ", m=n="<< s_max_m_n << ", k="<<s_max_k<< std::endl;
    cudaErrCheck(hipDeviceReset());
    }

    // for pressure test
    if (precision == "FP16_32_TENSOR" && perf == false) {
    std::cout << "[TensorCore FP16(FP32 accumulation) Time and TFLOPS Result]" << std::endl;
    std::cout << std::setw(7) << "m" << std::setw(7) << "n" << std::setw(7) << "k";
    std::cout << std::setw(15) << "Time (msec)" << std::setw(15) << "TFLOPS";
    std::cout << std::endl;
    s_max_tflops = 0;
    numRepeats = 2000;
    std::vector<int> mnk={512, 1024, 5120, 10240};
    for(int i=0; i<mnk.size(); i++) calFP16Accu32Tensor( mnk[i], mnk[i], mnk[i], s_max_tflops, s_max_m_n, s_max_k, numRepeats);
    cudaErrCheck(hipDeviceReset());
    }
    
    //======= for cudacore test
    if (precision == "FP32_CUDA" && perf == true) {
    std::cout << "[CUDA core FP32 Time and TFLOPS Result]" << std::endl;
    std::cout << std::setw(7) << "m" << std::setw(7) << "n" << std::setw(7) << "k";
    std::cout << std::setw(15) << "Time (msec)" << std::setw(15) << "TFLOPS";
    std::cout << std::endl;
    s_max_tflops = 0;
    numRepeats = 10;
    for(m=1024, n = 1024; m <= 25600; m+=4096, n+=4096) {
    for(k=1024; k <= 20480; k+=4096) {
	calFP32CUDA( m, n, k,s_max_tflops, s_max_m_n, s_max_k, numRepeats);
    }}
    std::cout << "[Peak TFLOPS]=" << s_max_tflops << ", m=n="<< s_max_m_n << ", k="<<s_max_k<< std::endl;
    cudaErrCheck(hipDeviceReset());
    }
    // for pressure test
    if (precision == "FP32_CUDA" && perf == false) {
    std::cout << "[CUDA core FP32 Time and TFLOPS Result]" << std::endl;
    std::cout << std::setw(7) << "m" << std::setw(7) << "n" << std::setw(7) << "k";
    std::cout << std::setw(15) << "Time (msec)" << std::setw(15) << "TFLOPS";
    std::cout << std::endl;
    s_max_tflops = 0;
    numRepeats = 2000;
    std::vector<int> mnk={512, 1024, 5120, 10240};
    for(int i=0; i<mnk.size(); i++) calFP32CUDA( mnk[i], mnk[i], mnk[i], s_max_tflops, s_max_m_n, s_max_k, numRepeats);
    cudaErrCheck(hipDeviceReset());
    }
    // for perf test
    if (precision == "FP16_CUDA" && perf == true) {
    std::cout << "[CUDA core FP16 Time and TFLOPS Result]" << std::endl;
    std::cout << std::setw(7) << "m" << std::setw(7) << "n" << std::setw(7) << "k";
    std::cout << std::setw(15) << "Time (msec)" << std::setw(15) << "TFLOPS";
    std::cout << std::endl;
    s_max_tflops = 0;
    numRepeats = 10;
    for(m=1024, n = 1024; m <= 25600; m+=4096, n+=4096) {
    for(k=1024; k <= 20480; k+=4096) {
	calFP16CUDA( m, n, k,s_max_tflops, s_max_m_n, s_max_k, numRepeats);
    }}
    std::cout << "[Peak TFLOPS]=" << s_max_tflops << ", m=n="<< s_max_m_n << ", k="<<s_max_k<< std::endl;
    cudaErrCheck(hipDeviceReset());
    }
    // for pressure test
    if (precision == "FP16_CUDA" && perf == false) {
    std::cout << "[CUDA core FP16 Time and TFLOPS Result]" << std::endl;
    std::cout << std::setw(7) << "m" << std::setw(7) << "n" << std::setw(7) << "k";
    std::cout << std::setw(15) << "Time (msec)" << std::setw(15) << "TFLOPS";
    std::cout << std::endl;
    s_max_tflops = 0;
    numRepeats = 2000;
    std::vector<int> mnk={512, 1024, 5120, 10240};
    for(int i=0; i<mnk.size(); i++) calFP16CUDA( mnk[i], mnk[i], mnk[i], s_max_tflops, s_max_m_n, s_max_k, numRepeats);
    cudaErrCheck(hipDeviceReset());
    }

    return 0;
}

